// 两个向量加法kernel，grid和block均为一维
__global__ void add(float* x, float * y, float* z, int n)
{
    // 获取该线程的全局索引
    int index = threadIdx.x + blockIdx.x * blockDim.x;
    // 步长(线程总数)
    int stride = blockDim.x * gridDim.x;
    for (int i = index; i < n; i += stride)
    {
        z[i] = x[i] + y[i];
    }
}

#include <iostream>
#include "hip/hip_runtime.h"  

void lod_weight(float* dst, float* src, int n, hipStream_t stream) {
  hipMemcpyAsync((void*)dst, (void*)src, n, hipMemcpyHostToDevice, stream);
}

int main()
{
    int N = 1 << 20; //(2^20)
    int nBytes = N * sizeof(float); // 2^20 * 4 = 4 MiB 
    
    // 申请 Host 内存，存放输入、输出和 权重数据
    float *input, *output, *weight_0, *weight_1, *weight_2, *weight_3;
    input = (float*)malloc(nBytes);
    output = (float*)malloc(nBytes);
    weight_0 = (float*)malloc(nBytes);
    // weight_1 = (float*)malloc(nBytes);
    // weight_2 = (float*)malloc(nBytes);
    // weight_3 = (float*)malloc(nBytes);

    hipHostMalloc(&weight_1, nBytes, hipHostMallocDefault);
    hipHostMalloc(&weight_2, nBytes, hipHostMallocDefault);
    hipHostMalloc(&weight_3, nBytes, hipHostMallocDefault);

    // 初始化输入数据和权重数据
    for (int i = 0; i < N; ++i){
      input[i] = 10.0;
      weight_0[i] = weight_1[i] = weight_2[i] = weight_3[i] = 20.0;
    }

    // 创建一个 CUDA kernel 计算流
    hipStream_t compute_stream;
    hipStreamCreateWithFlags(&compute_stream, hipStreamNonBlocking);

    // 创建 event
    hipEvent_t pre_compute_event;
    hipEvent_t compute_event;
    hipEvent_t load_event;
    hipEventCreate(&pre_compute_event);
    hipEventCreate(&compute_event);
    hipEventCreate(&load_event);

    // 申请 GPU 显存
    float *d_input, *d_output, *d_weight_0, *d_weight_1;
    hipMalloc((void**)&d_input, nBytes);
    hipMalloc((void**)&d_output, nBytes);
    hipMalloc((void**)&d_weight_0, nBytes);
    hipMalloc((void**)&d_weight_1, nBytes);

    // 这里只将输入数据和 weight_0 数据从 Host 拷贝到 GPU, weight_1, weight_2, weight_3在执行时按需加载
    hipMemcpy((void*)d_input, (void*)input, nBytes, hipMemcpyHostToDevice);
    hipMemcpy((void*)d_weight_0, (void*)weight_0, nBytes, hipMemcpyHostToDevice);
    
    // 定义kernel的执行配置
    dim3 blockSize(1024);
    dim3 gridSize(16);

    // 执行 kernel 和 数据加载，并做好同步, kernel 在 计算流上， 数据加载在空流上
    add <<< gridSize, blockSize, 0, compute_stream >>>(d_input, d_weight_0, d_output, N);    
    hipEventRecord(pre_compute_event, compute_stream);

    lod_weight(d_weight_1, weight_1, nBytes, NULL);
    hipEventRecord(load_event, NULL);

    hipStreamWaitEvent(compute_stream, load_event);
    add <<< gridSize, blockSize, 0, compute_stream >>>(d_output, d_weight_1, d_input, N);
    hipEventRecord(compute_event, compute_stream);
    
    hipStreamWaitEvent(NULL, pre_compute_event);
    lod_weight(d_weight_0, weight_2, nBytes, NULL);
    hipEventRecord(load_event, NULL);

    hipStreamWaitEvent(compute_stream, load_event);
    add <<< gridSize, blockSize, 0, compute_stream >>>(d_input, d_weight_0, d_output, N);

    hipStreamWaitEvent(NULL, compute_event);
    lod_weight(d_weight_1, weight_3, nBytes, NULL);
    hipEventRecord(load_event, NULL);

    hipStreamWaitEvent(compute_stream, load_event);
    add <<< gridSize, blockSize, 0, compute_stream >>>(d_output, d_weight_1, d_input, N);
    
    // 流同步
    hipStreamSynchronize(NULL);
    hipStreamSynchronize(compute_stream);
    
    // 将device得到的结果拷贝到host(注意 最后的结果存在了 d_input 中)
    hipMemcpy((void*)output, (void*)d_input, nBytes, hipMemcpyDeviceToHost);
    
    // 检查执行结果
    float maxError = 0.0;
    for (int i = 0; i < N; i++)
        maxError = fmax(maxError, fabs(output[i] - 90.0));
    std::cout << "最大误差: " << maxError << std::endl;

    // 释放 GPU 显存
    hipFree(d_input);
    hipFree(d_output);
    hipFree(d_weight_0);
    hipFree(d_weight_1);

    // 释放 Host 内存
    free(input);
    free(output);
    free(weight_0);
    // free(weight_1);
    // free(weight_2);
    // free(weight_3);
    hipHostFree(weight_1);
    hipHostFree(weight_2);
    hipHostFree(weight_3);

    // 释放cuda stream
    hipStreamDestroy(compute_stream);
    
    // 释放 event
    hipEventDestroy(compute_event);
    hipEventDestroy(pre_compute_event);
    hipEventDestroy(load_event);
    return 0;
}
